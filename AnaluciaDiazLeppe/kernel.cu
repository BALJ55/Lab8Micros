#include "hip/hip_runtime.h"
#include "kernel.h"

#include <stdio.h>


#define TX 32

#define TY 32

#define RAD 1


/* 
int divUp(int a, int b){

	return (a + b - 1)/b;

}

*/
// clip values to [0 , 255]

__device__ unsigned char clip(int n){

  return n > 255 ? 255 : (n < 0 ? 0 : n);

}


// bound index values to max size

__device__ int idxClip(int idx, int idxMax){

 return idx >(idxMax - 1) ? (idxMax - 1) : (idx < 0 ? 0 : idx);

}


// transform img(c,r) to flat index i

__device__ int flatten(int col, int row, int width, int height) {

  return idxClip(col, width) + idxClip(row, height)*width;

}



__global__ void
filter_kernel(unsigned char *d_input, unsigned char *d_output, int rows, int cols,float* d_window, int window_size) {


  //define image row, col position

  const int c = threadIdx.x + blockDim.x * blockIdx.x;

  const int r = threadIdx.y + blockDim.y * blockIdx.y;
  // exit if out of image bounds

  if((c >= cols) || (r >= rows)) return;


  // compute flat index

  const int i = flatten(c, r, cols, rows);


  const int gloc= threadIdx.x + RAD;
  extern_shared_float data[];

  data[gloc]= d_input[i];

  if (threadIdx.x <RAD){
      data[gloc -RAD]-d_input[i-RAD];
      data[gloc +blockDim.x] - d_input[i +blockDim.x];
  }
  _syncthreads();

  float pixel_result = 0;
  //se aplica la convolucion a la imagen para esto se llegara a utilizar un for 
  for(int rd = -RAD; rd<= RAD; ++rd){
   for(int cd = -RAD; cd<= RAD; ++cd){
  //compute image and windows indexes
  int imgIdx= flatten(c +cd, r +rd, cols, rows);
  int fltIdx= flatten(RAD + cd, RAD +rd, window_size);  
  // lee la funcion del pixel y los indexes de la ventana
  uchar pixel_val =i_input[imgIdx];
  float weight = d_window[fltIdx];
  //se acumula el valor
  pixel_result += pixel_val * weight;
 }
}
//bound pixel value to [0,255] and store in output
d_output[i] = clip((uchar) pixel_result);
}

//definicion del wrapper function
void filter_gpu(Mat input, Mat output){
  //convolucion del window size
  const int window_size = 2 * RAD +1;
  const float edgeDetected[9] = {-1, -1, -1, -1, 8, -1, -1, -1, -1};
  unsigned char *inputPtr = (unsigned char*) input.data;
  unsigned char *outputPtr = (unsigned char*) output.data;
  unsigned char rows = input.rows;
  unsigned char cols= input.cols;
  //grid size dimensions (blocks)
  int Bx = (TX +cols -1)/TX;
  int By = (TY +rows -1)TY;
// se declaran punteros para memoria 
  unsigned char *d_in = 0;
  unsigned char *d_out= 0;
  float *d_window = 0;
//input y output en el device
  hipMalloc(&d_in, cols*rows*sizeof(unsigned char));
  hipMalloc(&d_out, cols*rows* sizeof(unsigned char)); 
  hipMalloc(&d_window, window_size*window_size * sizeof(float));

//de ejemplos en clase, del host al device
  hipMemcpy(d_in, inputPtr, cols*rows*sizeof(unsigned char), hipMemcpyHostToDevice);
  hipMemcpy(d_window, edgeDetect, window_size*window_size*sizeof(float),  cudaMemcpyHosttoDevice);
//dimensiones del kernel
  size_t sharedMemSize = (TX +2 *RAD) *sizeof(float);
  const dim3 blockSize = dim3(TX, TY);
  const dim3 gridSize = dim3(Bx, By);
//GPU y el kernel
  filter_kernel<<gridSize, blockSize, sharedMemSize >>> (d_in, d_out, rows, cols, d_window, window_size);
//copia del device al host
  hipMemcpy(outputPtr, d_out, rows*cols*sizeof(unsigned char) , hipMemcpyDeviceToHost);
  hipFree(d_in);
  hipFree(d_out);
  hipFree(d_window);
}
